#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i ,j , ld) (((j)*(ld))+(i))
#define n 5 // size of matrix A (i.e., number of column)
int main (void) {
    hipError_t cudaStat ; // cudaMalloc status
    hipblasStatus_t stat ; // CUBLAS functions status
    hipblasHandle_t handle ; // CUBLAS context
    int i, j; // i-row index , j-column index
    float *A; // A - n*n matrix on the host
    float *L; // L - n*n matrix on the host

    A = (float*) malloc(n * n * sizeof(float)); // host mem. alloc for A
    L = (float*) malloc(n * n * sizeof(float)); // host mem. alloc for L
    // define a positive definite matrix A
    A[0]  = 1; A[1]  = 2;  A[2]  = 4;   A[3]  = 3;  A[4]  = 2;
    A[5]  = 2; A[6]  = 13; A[7]  = 17;  A[8]  = 21; A[9]  = 10;
    A[10] = 4; A[11] = 17; A[12] = 29;  A[13] = 33; A[14] = 16;
    A[15] = 3; A[16] = 21; A[17] = 33;  A[18] = 68; A[19] = 24;
    A[20] = 2; A[21] = 10; A[22] = 16;  A[23] = 24; A[24] = 11;
    printf("Original matrix A:\n");
    for (i = 0; i < n; ++i){
        for (j = 0; j < n; ++j){
            printf(" %4.0f", A[IDX2C(i, j, n)]);
        }
        printf("\n");
    }
    float *d_L; // d_L - L on the device
    float sqrta11 = sqrt(A[0]);
    cudaStat = hipMalloc((void **)&d_L, n * n * sizeof(float)); 
    stat = hipblasCreate(&handle);
    stat = hipblasSetMatrix(n, n, sizeof(*L), A, n, d_L, n); // initialize d_L from A

    // compute first column
    stat = hipblasSscal(handle, n, &sqrta11, d_L, 1); 

    // compute second to last column
    float t1;
    float alpha = -1.0f;
    float beta = 1.0f;
    for (j = 1; j < n; ++j){
        stat = hipblasSgemv(
                handle, 
                HIPBLAS_OP_N, 
                n - j,   // number of rows of matrix
                j,       // number of cols of matrix 
                &alpha,  // alpha
                d_L + j, // pointer to matrix
                n,       // leading dimension of matrix 
                d_L + j, // pointer to x
                n,       // incx
                &beta,   // beta
                d_L + j * n + j, // pointer to y
                1);      // incy
        stat = hipblasGetVector(1, sizeof(float), d_L + j * n + j, 1, &t1, 1);
        t1 = 1 / sqrt(t1);
        stat = hipblasSscal(handle, n - j, &t1, d_L + j * n + j, 1);
    }

    // fetch result from GPU memory
    stat = hipblasGetMatrix(n, n, sizeof(float), d_L, n, L, n);

    printf("After cholesky decomposition:\n");
    for (i = 0; i < n; ++i){
        for (j = 0; j < n; ++j){
            printf(" %4.0f", L[IDX2C(i, j, n)]);
        }
        printf("\n");
    }

    hipFree(d_L); 
    hipblasDestroy(handle); 
    free(A); 
    free(L); 
    return EXIT_SUCCESS ;
}
