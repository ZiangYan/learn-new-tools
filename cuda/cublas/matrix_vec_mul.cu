#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i ,j , ld) (((j)*(ld))+(i))
#define m 6 // number of rows of a
#define n 5 // number of columns of a
int main (void) {
    hipError_t cudaStat ; // cudaMalloc status
    hipblasStatus_t stat ; // CUBLAS functions status
    hipblasHandle_t handle ; // CUBLAS context
    int i, j; // i-row index , j- column index
    float *a; // a -mxn matrix on the host
    float *x; // x - n- vector on the host
    float *y; // y - m- vector on the host
    a = (float*) malloc(m * n * sizeof(float)); // host mem. alloc for a
    x = (float*) malloc(n * sizeof (float)); // host mem. alloc for x
    y = (float*) malloc(m * sizeof (float)); // host mem. alloc for y
    // define an mxn matrix a - column by column
    int ind = 11; // a:
    for (j = 0; j < n; ++j){
        for (i = 0; i < m; ++i){
            a[IDX2C(i, j, m)] = (float)ind++;
        }
    }
    printf ("a:\n");
    for (i = 0; i < m; ++i){
        for (j = 0; j < n; ++j){
            printf(" %4.0f", a[IDX2C(i, j, m)]);
        }
        printf("\n");
    }
    for (i = 0; i < n; ++i) x[i] = 1.0f;
    for (j = 0; j < m; ++j) y[j] = 0.0f;
    // on the device
    float *d_a; // d_a - a on the device
    float *d_x; // d_x - x on the device
    float *d_y; // d_y - y on the device
    cudaStat = hipMalloc((void **)&d_a, m * n * sizeof(*a)); // device
    // memory alloc for a
    cudaStat = hipMalloc((void **)&d_x, n * sizeof(*x)); // device
    // memory alloc for x
    cudaStat = hipMalloc((void **)&d_y, m * sizeof(*y)); // device
    // memory alloc for y
    stat = hipblasCreate(&handle);
    stat = hipblasSetMatrix(m, n, sizeof(*a), a, m, d_a, m); // cp a- >d_a
    stat = hipblasSetVector(n, sizeof(*x),x, 1, d_x, 1); // cp x- >d_x
    stat = hipblasSetVector(m, sizeof(*y),y, 1, d_y, 1); // cp y- >d_y
    float al = 1.0f; // al =1
    float bet = 1.0f; // bet =1
    // matrix - vector multiplication : d_y = al*d_a *d_x + bet *d_y
    // d_a - mxn matrix ; d_x - n-vector , d_y - m- vector ;
    // al ,bet - scalars
    stat = hipblasSgemv(handle,HIPBLAS_OP_N, m, n, &al, d_a, m, d_x, 1, &bet, d_y, 1);
    stat = hipblasGetVector(m, sizeof(*y), d_y, 1, y, 1); // copy d_y - >y
    printf ("y after Sgemv ::\n");
    for(j = 0; j < m; j++) {
        printf(" %5.0f", y[j]); // print y after Sgemv
        printf("\n");
    }
    hipFree(d_a); // free device memory
    hipFree(d_x); // free device memory
    hipFree(d_y); // free device memory
    hipblasDestroy(handle); // destroy CUBLAS context
    free(a); // free host memory
    free(x); // free host memory
    free(y); // free host memory
    return EXIT_SUCCESS ;
}
