
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printid(){
    int id = blockIdx.x;
    printf("I'm a thread in block %d!\n", id);
    __syncthreads();
    
}


int main(int argc, char **argv){
    printid<<<16, 1>>>();
    hipDeviceSynchronize();
    printf("That's all!\n");
    return 0;
}
