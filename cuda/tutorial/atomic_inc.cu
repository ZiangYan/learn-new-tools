
#include <hip/hip_runtime.h>
#include <stdio.h>

const int ARRAY_SIZE = 10;
const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
const int NUM_THREAD = 100000;
const int BLOCK_WIDTH = 100;

__global__ void naive_inc(int *g){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx = idx % ARRAY_SIZE;
    g[idx] += 1;
}

__global__ void atomic_inc(int *g){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx = idx % ARRAY_SIZE;
    atomicAdd(g + idx, 1);
}

int main(int argc, char **argv){
    int * d_arr;
    hipMalloc((void **) &d_arr, ARRAY_BYTES);
    hipMemset((void *) d_arr, 0, ARRAY_BYTES);
    //naive_inc<<<NUM_THREAD / BLOCK_WIDTH, BLOCK_WIDTH>>>(d_arr);
    atomic_inc<<<NUM_THREAD / BLOCK_WIDTH, BLOCK_WIDTH>>>(d_arr);
    int h_arr[ARRAY_SIZE];
    hipMemcpy(h_arr, d_arr, ARRAY_BYTES, hipMemcpyDeviceToHost);
    for (int i = 0; i < ARRAY_SIZE; ++i){
        printf("%d \t", h_arr[i]);
    }
    return 0;
}

