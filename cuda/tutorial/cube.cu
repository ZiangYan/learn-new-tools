
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cube(float *d_in, float *d_out){
    int idx = threadIdx.x;
    float data = d_in[idx];
    d_out[idx] = data * data * data;
}


int main(int argc, char ** argv){
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    float h_in[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; ++i){
        h_in[i] = i;
    }
    float *d_in;
    float *d_out;
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    cube<<<1, ARRAY_SIZE>>>(d_in, d_out);
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    for (int i = 0; i < ARRAY_SIZE; ++i){
        printf("%f \t", h_out[i]);
        if (i % 4 == 0){
            printf("\n");
        }
    }
    return 0;
}

